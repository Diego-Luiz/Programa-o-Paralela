
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define max 1024 //Tamanho do Vetor
#define pi 3.1415 //numero PI com 4 casas decimais

/*Função que será executada pelos blocos da GPU e suas threads*/
__global__ void mpi(double *a, double *b){
      // id = id do bloco * dimensão do bloco * thread do block
      int id= blockIdx.x * blockDim.x + threadIdx.x;
    
      /*
      Verificação necessária para evitar acessos indevidos a memória
      */
      if(id< max) 
        b[id]= a[id]*pi; //realizando a operação de multiplicação
}

int main(){
    /*Variável para armazenar o tempo final gasto no processamento*/
    double time=0.0; 
    
    /*
      --> Como as variáveis estão sendo utilizadas:
      - h_a : ponteiro do tipo double que armazena o endereço do
      vetor na CPU e que será preenchido com seu índice de 
      posição + 1 posição
      - h_b : ponteiro do tipo double que armazena o endereço do
      vetor na CPU onde será guardado o resultado da operação de
      multiplicação do vetor h_a
      - d_a : ponteiro do tipo double que armazena o endereço do
      vetor na GPU e que receberá o vetor h_a pela função
      cudaMemcpy()
      - d_b : ponteiro do tipo double que armazena o endereço do
      vetor na GPU e que receberá o resultado da operação da
      multiplicação do vetor d_a
    */
    
    /*
        Variáveis do tipo double e ponteiro, que são usados pela CPU e
        que armazenam o endereço dos vetores que serão alocados 
    */
    
    double *h_a, *h_b;
    
    /*
        Alocando dinamicamente um vetor na memória de tamanho 
        max * tamanho de double
    */
    h_a= (double*) malloc(max*sizeof(double));
    
    /*
        Alocando dinamicamente um vetor na memória de tamanho 
        max * tamanho de double
    */
    h_b= (double*) malloc(max*sizeof(double));
    
    /*
        Laco que inicia cada posição da memória com seu
        respectivo índice + 1 unidade
    */
    for(int i=0;i<max;i++){
        h_a[i]= i+1;
    }
    /*
        Variáveis do tipo double e ponteiro que guardarão os
        endereços dos vetores na GPU 
    */
    double *d_a, *d_b;
    
    /*
        Alocando um vetor na memoria da GPU de tamanho 
        max * tamanho de double
    */
    hipMalloc(&d_a,max*sizeof(double));
    /*
        Alocando um vetor na memoria da GPU de tamanho 
        max * tamanho de double
    */
    hipMalloc(&d_b,max*sizeof(double));
    
    /*
      Função que copia o conteúdo do vetor h_a (CPU) para o vetor 
      d_a (CPU)
      Parâmetros: (destino, origem, tamanho do elemento a ser
      copiado, tipo de cópia)
    */
    hipMemcpy(d_a,h_a,max*sizeof(double),hipMemcpyHostToDevice);
    
    //---------------------------------------------------------------
    /*Variável do tipo clock_t, que inicia a contagem do tempo*/
    clock_t begin=clock(); 
    
    /*
      Invocando método responsável por realizar a operação
      Parâmetros: nome_função<<<numero de blocos, qtd
      threads>>>(argumento 1, argumento 2);
    */
    mpi<<<1024,1>>>(d_a,d_b);
    
    
    /*Variável do tipo clock_t, que termina a contagem do tempo*/
    clock_t end=clock(); 
    //---------------------------------------------------------------
    
    /*
      Função que copia o conteúdo do vetor d_b (GPU) para o vetor h_b
      (CPU)
      Parâmetros: (destino, origem, tamanho do elemento a ser
      copiado, tipo de copia)
    */
    hipMemcpy(h_b, d_b, max * sizeof(double), hipMemcpyDeviceToHost);
    
    /*Calculando o tempo em segundos*/
    time+= (double)(end - begin) / CLOCKS_PER_SEC;
    
    /*Exibindo o resultado final*/
    printf("Tempo gasto: %f segundos", time); 
    
    /*Liberando a memória da GPU*/
    hipFree(d_a);
    hipFree(d_b);
    
    /*Liberando a memória da CPU*/
    free(h_a);
    free(h_b);
    
    
    return 0;
}